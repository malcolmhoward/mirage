
#include <hip/hip_runtime.h>
/*
 * cuda_color_correction.cu - CUDA Color Correction Implementation
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 * By contributing to this project, you agree to license your contributions
 * under the GPLv3 (or any later version) or any future licenses chosen by
 * the project author(s). Contributions include any modifications,
 * enhancements, or additions to the project. These contributions become
 * part of the project and are adopted by the project author(s).
 *
 * High-performance color correction for NoIR cameras using CUDA
 */

#ifdef USE_CUDA

#include "cuda_color_correction.h"
#include <cuda_runtime.h>
#include <stdio.h>

/* CUDA error checking macro */
#define CUDA_CHECK(call) do { \
   cudaError_t error = call; \
   if (error != cudaSuccess) { \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", \
              __FILE__, __LINE__, cudaGetErrorString(error)); \
      return -1; \
   } \
} while(0)

/* Block dimensions for kernel launch */
#define BLOCK_WIDTH  16
#define BLOCK_HEIGHT 16

/* Device constant memory for color matrix (faster access) */
__constant__ float d_colorMatrix[3][3];
__constant__ float d_colorOffset[3];

/**
 * CUDA kernel for color correction
 * Applies a 3x3 color matrix transformation to RGBA image
 */
__global__ void colorCorrectionKernel(
   unsigned char* input,
   unsigned char* output,
   int width,
   int height)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   
   if (x >= width || y >= height) return;
   
   int idx = (y * width + x) * 4; // RGBA format
   
   /* Read input pixel values */
   float r = input[idx + 0];
   float g = input[idx + 1];
   float b = input[idx + 2];
   unsigned char a = input[idx + 3];
   
   /* Apply color matrix transformation */
   float r_new = d_colorMatrix[0][0] * r + 
                 d_colorMatrix[0][1] * g + 
                 d_colorMatrix[0][2] * b + 
                 d_colorOffset[0];
                 
   float g_new = d_colorMatrix[1][0] * r + 
                 d_colorMatrix[1][1] * g + 
                 d_colorMatrix[1][2] * b + 
                 d_colorOffset[1];
                 
   float b_new = d_colorMatrix[2][0] * r + 
                 d_colorMatrix[2][1] * g + 
                 d_colorMatrix[2][2] * b + 
                 d_colorOffset[2];
   
   /* Clamp values to valid range [0, 255] */
   r_new = fminf(fmaxf(r_new, 0.0f), 255.0f);
   g_new = fminf(fmaxf(g_new, 0.0f), 255.0f);
   b_new = fminf(fmaxf(b_new, 0.0f), 255.0f);
   
   /* Write output pixel */
   output[idx + 0] = (unsigned char)r_new;
   output[idx + 1] = (unsigned char)g_new;
   output[idx + 2] = (unsigned char)b_new;
   output[idx + 3] = a; // Preserve alpha
}

/**
 * Optimized kernel using shared memory for coalesced access
 */
__global__ void colorCorrectionKernelOptimized(
   unsigned char* __restrict__ input,
   unsigned char* __restrict__ output,
   int width,
   int height)
{
   __shared__ uchar4 tile[BLOCK_HEIGHT][BLOCK_WIDTH];
   
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   
   /* Load data into shared memory */
   if (x < width && y < height) {
      int idx = y * width + x;
      tile[threadIdx.y][threadIdx.x] = ((uchar4*)input)[idx];
   }
   __syncthreads();
   
   if (x >= width || y >= height) return;
   
   /* Process from shared memory */
   uchar4 pixel = tile[threadIdx.y][threadIdx.x];
   float r = pixel.x;
   float g = pixel.y;
   float b = pixel.z;
   
   /* Apply transformation using constant memory matrices */
   float r_new = d_colorMatrix[0][0] * r + 
                 d_colorMatrix[0][1] * g + 
                 d_colorMatrix[0][2] * b + 
                 d_colorOffset[0];
                 
   float g_new = d_colorMatrix[1][0] * r + 
                 d_colorMatrix[1][1] * g + 
                 d_colorMatrix[1][2] * b + 
                 d_colorOffset[1];
                 
   float b_new = d_colorMatrix[2][0] * r + 
                 d_colorMatrix[2][1] * g + 
                 d_colorMatrix[2][2] * b + 
                 d_colorOffset[2];
   
   /* Write output with clamping */
   int idx = y * width + x;
   ((uchar4*)output)[idx] = make_uchar4(
      (unsigned char)fminf(fmaxf(r_new, 0.0f), 255.0f),
      (unsigned char)fminf(fmaxf(g_new, 0.0f), 255.0f),
      (unsigned char)fminf(fmaxf(b_new, 0.0f), 255.0f),
      pixel.w
   );
}

/* Initialize CUDA color correction module */
int cuda_color_init(void)
{
   /* Check for CUDA device */
   int deviceCount;
   CUDA_CHECK(cudaGetDeviceCount(&deviceCount));
   
   if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
      return -1;
   }
   
   /* Set device and print info */
   CUDA_CHECK(cudaSetDevice(0));
   
   cudaDeviceProp prop;
   CUDA_CHECK(cudaGetDeviceProperties(&prop, 0));
   printf("CUDA Color Correction initialized on %s\n", prop.name);
   
   return 0;
}

/* Cleanup CUDA resources */
void cuda_color_cleanup(void)
{
   cudaDeviceReset();
}

/* Apply color correction to device memory */
int cuda_apply_color_correction(
   unsigned char* d_input,
   unsigned char* d_output,
   int width,
   int height,
   const cuda_color_matrix_t* matrix)
{
   /* Copy matrix to constant memory */
   CUDA_CHECK(cudaMemcpyToSymbol(d_colorMatrix, matrix->m, 
                                  sizeof(float) * 9));
   CUDA_CHECK(cudaMemcpyToSymbol(d_colorOffset, matrix->off, 
                                  sizeof(float) * 3));
   
   /* Configure kernel launch parameters */
   dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT);
   dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                 (height + blockSize.y - 1) / blockSize.y);
   
   /* Launch optimized kernel */
   colorCorrectionKernelOptimized<<<gridSize, blockSize>>>(
      d_input, d_output, width, height);
   
   /* Check for kernel errors */
   CUDA_CHECK(cudaGetLastError());
   CUDA_CHECK(cudaDeviceSynchronize());
   
   return 0;
}

/* Apply color correction to host memory */
int cuda_apply_color_correction_host(
   unsigned char* h_input,
   unsigned char* h_output,
   int width,
   int height,
   const cuda_color_matrix_t* matrix)
{
   size_t imageSize = width * height * 4 * sizeof(unsigned char);
   unsigned char *d_input, *d_output;
   
   /* Allocate device memory */
   CUDA_CHECK(cudaMalloc(&d_input, imageSize));
   CUDA_CHECK(cudaMalloc(&d_output, imageSize));
   
   /* Copy input to device */
   CUDA_CHECK(cudaMemcpy(d_input, h_input, imageSize, 
                         cudaMemcpyHostToDevice));
   
   /* Apply color correction */
   int result = cuda_apply_color_correction(d_input, d_output, 
                                            width, height, matrix);
   
   if (result == 0) {
      /* Copy result back to host */
      CUDA_CHECK(cudaMemcpy(h_output, d_output, imageSize, 
                            cudaMemcpyDeviceToHost));
   }
   
   /* Free device memory */
   cudaFree(d_input);
   cudaFree(d_output);
   
   return result;
}

/* Optimized version using pre-allocated device memory */
int cuda_apply_color_correction_optimized(
   unsigned char* h_input,
   unsigned char* h_output,
   unsigned char* d_buffer,
   int width,
   int height,
   const cuda_color_matrix_t* matrix,
   cudaStream_t stream)
{
   size_t imageSize = width * height * 4 * sizeof(unsigned char);
   
   /* Copy matrix to constant memory */
   CUDA_CHECK(cudaMemcpyToSymbolAsync(d_colorMatrix, matrix->m, 
                                       sizeof(float) * 9, 0,
                                       cudaMemcpyHostToDevice, stream));
   CUDA_CHECK(cudaMemcpyToSymbolAsync(d_colorOffset, matrix->off, 
                                       sizeof(float) * 3, 0,
                                       cudaMemcpyHostToDevice, stream));
   
   /* Copy input to device using the pre-allocated buffer */
   CUDA_CHECK(cudaMemcpyAsync(d_buffer, h_input, imageSize,
                               cudaMemcpyHostToDevice, stream));
   
   /* Configure kernel launch parameters */
   dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT);
   dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                 (height + blockSize.y - 1) / blockSize.y);
   
   /* Launch kernel for in-place processing */
   colorCorrectionKernelOptimized<<<gridSize, blockSize, 0, stream>>>(
      d_buffer, d_buffer, width, height);
   
   /* Check for kernel errors */
   CUDA_CHECK(cudaGetLastError());
   
   /* Copy result back to host */
   CUDA_CHECK(cudaMemcpyAsync(h_output, d_buffer, imageSize,
                               cudaMemcpyDeviceToHost, stream));
   
   /* Synchronize stream */
   CUDA_CHECK(cudaStreamSynchronize(stream));
   
   return 0;
}

#endif /* USE_CUDA */

